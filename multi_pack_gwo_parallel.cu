#include <bits/stdc++.h>
#include <vector>
#include <iostream>
#include <stdio.h>
#include <stdlib.h>
#include <random>
#include <cmath>
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include "multi_pack_gwo_parallel_cuda.h"
# define PI	    3.14159265358979323846
# define CLIP   10
using namespace std;

int main(int argc, char *argv[]){
    
    hipEvent_t prog,start,end;

    hipEventCreate(&prog);
    hipEventCreate(&start);
    hipEventCreate(&end);

    hipEventRecord(prog,0);
    
    int num_packs = stoi(argv[1]);
    int wolves_per_pack = stoi(argv[2]);
    int num_iterations = stoi(argv[3]);
    int k = 3;
    int a = 2;

    float *wolves = (float *) malloc(sizeof(float) * num_packs * wolves_per_pack * 2);

    float *fitness_scores = (float *) malloc(sizeof(float) * num_packs * wolves_per_pack);

    float *guides = (float *) malloc(sizeof(float) * num_packs * k * 2);

    float *omega = (float *) malloc(sizeof(float) * num_packs * 2);

    float *sigma = (float *) malloc(sizeof(float) * num_packs * 2);

    for(int pack = 0; pack < num_packs; pack++){
        for(int wolf = 0; wolf < wolves_per_pack; wolf++){
            wolves[pack * wolves_per_pack * 2 + wolf * 2] = get_rand_num(-10,10);
            wolves[pack * wolves_per_pack * 2 + wolf * 2 + 1] = get_rand_num(-10,10);
        }
    }

    float *d_wolves, *d_fitness_scores, *d_guides, *d_omega, *d_sigma;

    hipMalloc(&d_wolves, sizeof(float) * num_packs * wolves_per_pack * 2);
    hipMalloc(&d_fitness_scores, sizeof(float) * num_packs * wolves_per_pack);
    hipMalloc(&d_guides,sizeof(float) * num_packs * k * 2);
    hipMalloc(&d_omega,sizeof(float) * num_packs * 2);
    hipMalloc(&d_sigma,sizeof(float) * num_packs * 2);

    hipMemcpy(d_wolves,wolves,sizeof(float) * num_packs * wolves_per_pack * 2,hipMemcpyHostToDevice);

    hipEventRecord(start,0);

    for(int iter = 0; iter < num_iterations; iter++){

        get_fitness<<<1,3 * 10>>>(d_wolves, d_fitness_scores, num_packs, wolves_per_pack, k);
        get_guides_omega_sigma<<<1,3 * 10>>>(d_wolves, d_fitness_scores, d_guides, d_omega, d_sigma, num_packs, wolves_per_pack, k);
        update<<<1,3 * 10>>>(d_wolves,d_guides,d_omega,d_sigma,num_packs,wolves_per_pack,k,a,get_rand_num(0,100));

    }

    hipMemcpy(guides,d_guides,sizeof(float) * num_packs * k * 2,hipMemcpyDeviceToHost);

    hipEventRecord(end,0);

    float prog_time, par_time;
`
    hipEventSynchronize(end);

    hipEventElapsedTime(&prog_time,prog,end);
    hipEventElapsedTime(&par_time,start,end);

    printf("%d,%d,%d,%f,%f\n", num_packs, wolves_per_pack, num_iterations,prog_time,par_time);

    // for(int i = 0; i<1; i++){
    //     for(int j = 0; j < k; j++){
    //         printf("%f %f\n",guides[i * k * 2 + j],guides[i * k * 2 + j + 1]);
    //     }
    // }
}   